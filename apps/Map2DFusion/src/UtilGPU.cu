
#include <hip/hip_runtime.h>
/******************************************************************************

  This file is part of Map2DFusion.

  Copyright 2016 (c)  Yong Zhao <zd5945@126.com> http://www.zhaoyong.adv-ci.com

  ----------------------------------------------------------------------------

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program. If not, see <http://www.gnu.org/licenses/>.

*******************************************************************************/
#ifdef HAS_CUDA

#include "UtilGPU.cuh"
#include <stdio.h>
#include <base/time/Global_Timer.h>

__global__ void pyrDown(float4* in_data,int in_rows,int in_cols,float4* out_data,int out_rows,int out_cols)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < out_rows &&  x< out_cols)
    {
        float4* in_ptr=in_data+(y<<2)*out_cols+(x<<1);//in_data+y*2*in_cols+2*x=in_data[2*y][2*x]
        out_data[x+y*out_cols].x=0.25*(in_ptr[0].x+in_ptr[1].x+in_ptr[in_cols].x+in_ptr[in_cols+1].x);
        out_data[x+y*out_cols].y=0.25*(in_ptr[0].y+in_ptr[1].y+in_ptr[in_cols].y+in_ptr[in_cols+1].y);
        out_data[x+y*out_cols].z=0.25*(in_ptr[0].z+in_ptr[1].z+in_ptr[in_cols].z+in_ptr[in_cols+1].z);
        out_data[x+y*out_cols].w=0.25*(in_ptr[0].w+in_ptr[1].w+in_ptr[in_cols].w+in_ptr[in_cols+1].w);
    }
}

__global__ void pyrDownArray(float4** in_data,int in_rows,int in_cols,float4** out_datas,int out_rows,int out_cols,int num)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < out_rows &&  x< out_cols)
    {
        for(int i=0;i<num;i++)
        {
            float4* in_ptr=in_data[i]+(y<<2)*out_cols+(x<<1);//in_data+y*2*in_cols+2*x=in_data[2*y][2*x]
            float4* out_ptr=out_datas[i]+x+y*out_cols;
            (*out_ptr).x=0.25*(in_ptr[0].x+in_ptr[1].x+in_ptr[in_cols].x+in_ptr[in_cols+1].x);
            (*out_ptr).y=0.25*(in_ptr[0].y+in_ptr[1].y+in_ptr[in_cols].y+in_ptr[in_cols+1].y);
            (*out_ptr).z=0.25*(in_ptr[0].z+in_ptr[1].z+in_ptr[in_cols].z+in_ptr[in_cols+1].z);
            (*out_ptr).w=0.25*(in_ptr[0].w+in_ptr[1].w+in_ptr[in_cols].w+in_ptr[in_cols+1].w);
        }
    }
}

__global__ void pyrUp(float4* in_data,int in_rows,int in_cols,float4* out_data,int out_rows,int out_cols)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < out_rows &&  x< out_cols)
    {
        float4* in_ptr=in_data+(y>>1)*in_cols+(x<<1);//in_data+y*2*in_cols+2*x=in_data[2*y][2*x]
        out_data[x+y*out_cols].x=0.25*(in_ptr[0].x+in_ptr[1].x+in_ptr[in_cols].x+in_ptr[in_cols+1].x);
        out_data[x+y*out_cols].y=0.25*(in_ptr[0].y+in_ptr[1].y+in_ptr[in_cols].y+in_ptr[in_cols+1].y);
        out_data[x+y*out_cols].z=0.25*(in_ptr[0].z+in_ptr[1].z+in_ptr[in_cols].z+in_ptr[in_cols+1].z);
        out_data[x+y*out_cols].w=0.25*(in_ptr[0].w+in_ptr[1].w+in_ptr[in_cols].w+in_ptr[in_cols+1].w);
    }
}

__global__ void pyrUpArray(float4** in_data,int in_rows,int in_cols,float4** out_datas,int out_rows,int out_cols,int num)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < out_rows &&  x< out_cols)
    {
        for(int i=0;i<num;i++)
        {
            float4* in_ptr=in_data[i]+(y<<2)*out_cols+(x<<1);//in_data+y*2*in_cols+2*x=in_data[2*y][2*x]
            float4* out_ptr=out_datas[i]+x+y*out_cols;
            (*out_ptr).x=0.25*(in_ptr[0].x+in_ptr[1].x+in_ptr[in_cols].x+in_ptr[in_cols+1].x);
            (*out_ptr).y=0.25*(in_ptr[0].y+in_ptr[1].y+in_ptr[in_cols].y+in_ptr[in_cols+1].y);
            (*out_ptr).z=0.25*(in_ptr[0].z+in_ptr[1].z+in_ptr[in_cols].z+in_ptr[in_cols+1].z);
            (*out_ptr).w=0.25*(in_ptr[0].w+in_ptr[1].w+in_ptr[in_cols].w+in_ptr[in_cols+1].w);
        }
    }
}

template <class T>
__global__ void warpPerspectiveKernel(int in_rows,int in_cols,T* in_data,
                                      int out_rows,int out_cols,T* out_data,
                                      float* inv,T defVar)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < out_rows &&  x< out_cols)
    {
        float srcX=inv[0]*x+inv[1]*y+inv[2];
        float srcY=inv[3]*x+inv[4]*y+inv[5];
        float srcW=inv[6]*x+inv[7]*y+inv[8];
        srcW=1./srcW;srcX*=srcW;srcY*=srcW;
        if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
        {
            out_data[x+y*out_cols]=in_data[(int)srcX+((int)srcY)*in_cols];
        }
        else
        {
            out_data[x+y*out_cols]=defVar;
        }
    }
}

template <class T>
bool operate<T>::warpPerspectiveCaller(int in_rows,int in_cols,T* in_data,
                           int out_rows,int out_cols,T* out_data,
                           float* inv,T defVar)
{
    T* in_dataGPU;
    T* out_dataGPU;
    float* invGPU;
    cudaMalloc((void**) &in_dataGPU, in_cols*in_rows*sizeof(T));
    cudaMalloc((void**) &out_dataGPU,out_cols*out_rows*sizeof(T));
    cudaMalloc((void**) &invGPU,9*sizeof(float));
    cudaMemcpy(in_dataGPU,in_data,in_cols*in_rows*sizeof(T),cudaMemcpyHostToDevice);
    cudaMemcpy(invGPU,inv,9*sizeof(float),cudaMemcpyHostToDevice);

    dim3 threads(32,32);
        dim3 grid(divUp(out_cols, threads.x), divUp(out_rows, threads.y));
//        dim3 grid(20,20);

    pi::timer.enter("warpPerspectiveKernel");
    warpPerspectiveKernel<T><<<grid,threads>>>(in_rows,in_cols,in_dataGPU,
                                               out_rows,out_cols,out_dataGPU,
                                               invGPU,defVar);
    pi::timer.leave("warpPerspectiveKernel");

    cudaMemcpy(out_data,out_dataGPU,out_cols*out_rows*sizeof(T),cudaMemcpyDeviceToHost);
    cudaFree(in_dataGPU);cudaFree(out_dataGPU);cudaFree(invGPU);
    return true;
}

bool warpPerspective_uchar1(int in_rows,int in_cols,uchar1* in_data,
                            int out_rows,int out_cols,uchar1* out_data,
                            float* inv,uchar1 defVar)
{
    return operate<uchar1>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}
//bool warpPerspective_uchar2();
bool warpPerspective_uchar3(int in_rows,int in_cols,uchar3* in_data,
                            int out_rows,int out_cols,uchar3* out_data,
                            float* inv,uchar3 defVar)
{
    return operate<uchar3>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}

bool warpPerspective_uchar4(int in_rows,int in_cols,uchar4* in_data,
                            int out_rows,int out_cols,uchar4* out_data,
                            float* inv,uchar4 defVar)
{
    return operate<uchar4>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}


__global__ void renderFrameKernel(int in_rows,int in_cols,uchar3* in_data,//image in
                                  int out_rows,int out_cols,uchar4* out_data,
                                  bool fresh,uchar4 defVar,//image out
                                  float* inv,int centerX,int centerY//relations
                                  )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(y<out_rows&&x<out_cols)
    {
        int idxOut=x+y*out_cols;

        // find source location
        float srcX=inv[0]*x+inv[1]*y+inv[2];
        float srcY=inv[3]*x+inv[4]*y+inv[5];
        float srcW=inv[6]*x+inv[7]*y+inv[8];
        srcW=1./srcW;srcX*=srcW;srcY*=srcW;

        if(fresh) //warp
        {
            if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
            {
                uchar4* ptrOut=out_data+idxOut;
                *((uchar3*)ptrOut)=in_data[(int)srcX+((int)srcY)*in_cols];
                // compute weight
                {
                    //image weight
                    float difX=srcX-in_rows*0.5;
                    float difY=srcY-in_cols*0.5;
                    srcW=1000*(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols);
                    if(srcW<1) srcW=1;
                    //center weight
                }
                ptrOut->w=srcW;
            }
            else
            {
                out_data[idxOut]=defVar;
            }

        }
        else if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)// blender
        {
            uchar4* ptrOut=out_data+idxOut;
            // compute weight
            {
                //image weight
                float difX=srcX-in_rows*0.5;
                float difY=srcY-in_cols*0.5;
                srcW=1000*(0.25-(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols));
                if(srcW<1) srcW=1;
                //center weight
            }
            if(ptrOut->w<srcW)
            {
                ptrOut->w=srcW;
                uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                *((uchar3*)ptrOut)=*ptrIn;
            }
        }
    }
}

__global__ void renderFramesKernel(int in_rows,int in_cols,uchar3* in_data,//image in
                                   int out_rows,int out_cols,uchar4** out_datas,
                                   bool* freshs,uchar4 defVar,//image out
                                   float* invs,int* centers,int eleNum//relations
                                  )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(y<out_rows&&x<out_cols)
    {
        for(int i=0;i<eleNum;i++)
        {
            uchar4* out_data=out_datas[i];
            bool    fresh=freshs[i];
            float*  inv=invs+9*i;

            int idxOut=x+y*out_cols;
            uchar4* ptrOut=out_data+idxOut;

            // find source location
            float srcX=inv[0]*x+inv[1]*y+inv[2];
            float srcY=inv[3]*x+inv[4]*y+inv[5];
            float srcW=inv[6]*x+inv[7]*y+inv[8];
            srcW=1./srcW;srcX*=srcW;srcY*=srcW;

            if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
            {
                // compute weight
                {
                    //image weight
                    float difX=srcX-in_rows*0.5;
                    float difY=srcY-in_cols*0.5;
                    srcW=(0.25-(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols));//0~0.25
                    //center weight
                    if(1)
                    {
                        difX=centers[i*2]-x;
                        difY=centers[i*2+1]-y;
                        srcW=5e4*srcW/sqrt(difX*difX+difY*difY+1);
                    }
                    else
                        srcW=1000*srcW;
                    if(srcW<1) srcW=1;
                    else if(srcW>255) srcW=255;
                }
                if(fresh)
                {
                    *((uchar3*)ptrOut)=in_data[(int)srcX+((int)srcY)*in_cols];

                    ptrOut->w=srcW;
                }
                else// blender
                {
                    if(ptrOut->w<srcW)
                    {
                        ptrOut->w=srcW;
                        uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                        *((uchar3*)ptrOut)=*ptrIn;
                    }
                }
            }
            else if(fresh)
            {
                *ptrOut=defVar;
            }
        }
    }
}

__global__ void renderFramesKernel(int in_rows,int in_cols,uchar3* in_data,//image in
                                   int out_rows,int out_cols,float4** out_datas,
                                   bool* freshs,float4 defVar,//image out
                                   float* invs,int* centers,int eleNum//relations
                                  )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(y<out_rows&&x<out_cols)
    {
        for(int i=0;i<eleNum;i++)
        {
            float4* out_data=out_datas[i];
            bool    fresh=freshs[i];
            float*  inv=invs+9*i;

            int idxOut=x+y*out_cols;
            float4* ptrOut=out_data+idxOut;

            // find source location
            float srcX=inv[0]*x+inv[1]*y+inv[2];
            float srcY=inv[3]*x+inv[4]*y+inv[5];
            float srcW=inv[6]*x+inv[7]*y+inv[8];
            srcW=1./srcW;srcX*=srcW;srcY*=srcW;

            if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
            {
                // compute weight
                {
                    if(0)
                    {
                        //image weight //1-sqrt(dis)/dis_max;
                        float difX=srcX/(float)in_cols-0.5;
                        float difY=srcY/(float)in_rows-0.5;
                        srcW=0.5-(difX*difX+difY*difY);//0~0.25
                    }
                    else srcW=1;

                    //center weight
                    if(1)
                    {
                        float difX=centers[i*2]-x;
                        float difY=centers[i*2+1]-y;
                        srcW=1e5*srcW/(difX*difX+difY*difY+1000);
                    }
                }
                if(fresh)
                {
                    uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                    ptrOut->x=ptrIn->x*0.00392f;//~=/256
                    ptrOut->y=ptrIn->y*0.00392f;
                    ptrOut->z=ptrIn->z*0.00392f;
                    ptrOut->w=srcW;
                }
                else if(ptrOut->w<=srcW)
                {
                    uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                    float   sumweightInv=1./(ptrOut->w*2+srcW);
                    ptrOut->x=(ptrOut->x*ptrOut->w*2+ptrIn->x*0.00392f*srcW)*sumweightInv;//~=/256
                    ptrOut->y=(ptrOut->y*ptrOut->w*2+ptrIn->y*0.00392f*srcW)*sumweightInv;
                    ptrOut->z=(ptrOut->z*ptrOut->w*2+ptrIn->z*0.00392f*srcW)*sumweightInv;
                    ptrOut->w=srcW;
                }
            }
            else if(fresh)
            {
                *ptrOut=defVar;
            }
        }
    }
}

bool renderFrameCaller(CudaImage<uchar3>& rgbIn,CudaImage<uchar4>& ele,
                       float* inv,int centerX,int centerY)
{
    float* invGPU;
    checkCudaErrors(cudaMalloc((void**) &invGPU,9*sizeof(float)));
    checkCudaErrors(cudaMemcpy(invGPU,inv,9*sizeof(float),cudaMemcpyHostToDevice));
    dim3 threads(32,32);
    uchar4 defVar;
    defVar.x=defVar.y=defVar.z=defVar.w=0;
    dim3 grid(divUp(ele.cols, threads.x), divUp(ele.rows, threads.y));
    renderFrameKernel<<<grid,threads>>>(rgbIn.rows,rgbIn.cols,rgbIn.data,
                                        ele.rows,ele.cols,ele.data,
                                        ele.fresh,defVar,invGPU,centerX,centerY);
    checkCudaErrors(cudaFree(invGPU));
    return true;
}


bool renderFramesCaller(CudaImage<uchar3>& rgbIn,int out_rows,int out_cols,
                        uchar4** out_datas,bool* freshs,
                       float* invs,int* centers,int eleNum)
{
    float* invGPU;
    uchar4** outDataGPU;
    bool*  freshesGPU;
    int*   centersGPU;

    checkCudaErrors(cudaMalloc((void**) &invGPU,9*sizeof(float)*eleNum));
    checkCudaErrors(cudaMalloc((void**) &outDataGPU,sizeof(uchar4*)*eleNum));
    checkCudaErrors(cudaMalloc((void**) &freshesGPU,sizeof(bool)*eleNum));
    checkCudaErrors(cudaMalloc((void**) &centersGPU,2*sizeof(int)*eleNum));
    checkCudaErrors(cudaMemcpy(invGPU,invs,9*sizeof(float)*eleNum,cudaMemcpyHostToDevice));
    checkCudaErrors(cudaMemcpy(outDataGPU,out_datas,sizeof(uchar4*)*eleNum,cudaMemcpyHostToDevice));
    checkCudaErrors(cudaMemcpy(freshesGPU,freshs,sizeof(bool)*eleNum,cudaMemcpyHostToDevice));
    checkCudaErrors(cudaMemcpy(centersGPU,centers,2*sizeof(int)*eleNum,cudaMemcpyHostToDevice));
    dim3 threads(32,32);
    uchar4 defVar;
    defVar.x=defVar.y=defVar.z=defVar.w=0;
    dim3 grid(divUp(out_cols, threads.x), divUp(out_rows, threads.y));
    renderFramesKernel<<<grid,threads>>>(rgbIn.rows,rgbIn.cols,rgbIn.data,
                                        out_rows,out_cols,outDataGPU,freshesGPU,
                                        defVar,invGPU,centersGPU,eleNum);
    checkCudaErrors(cudaFree(invGPU));
    checkCudaErrors(cudaFree(outDataGPU));
    checkCudaErrors(cudaFree(freshesGPU));
    checkCudaErrors(cudaFree(centersGPU));
    return true;
}

bool renderFramesCaller(CudaImage<uchar3>& rgbIn,int out_rows,int out_cols,
                        float4** out_datas,bool* freshs,
                       float* invs,int* centers,int eleNum)
{
    float* invGPU;
    float4** outDataGPU;
    bool*  freshesGPU;
    int*   centersGPU;

    checkCudaErrors(cudaMalloc((void**) &invGPU,9*sizeof(float)*eleNum));
    checkCudaErrors(cudaMalloc((void**) &outDataGPU,sizeof(float4*)*eleNum));
    checkCudaErrors(cudaMalloc((void**) &freshesGPU,sizeof(bool)*eleNum));
    checkCudaErrors(cudaMalloc((void**) &centersGPU,2*sizeof(int)*eleNum));
    checkCudaErrors(cudaMemcpy(invGPU,invs,9*sizeof(float)*eleNum,cudaMemcpyHostToDevice));
    checkCudaErrors(cudaMemcpy(outDataGPU,out_datas,sizeof(float4*)*eleNum,cudaMemcpyHostToDevice));
    checkCudaErrors(cudaMemcpy(freshesGPU,freshs,sizeof(bool)*eleNum,cudaMemcpyHostToDevice));
    checkCudaErrors(cudaMemcpy(centersGPU,centers,2*sizeof(int)*eleNum,cudaMemcpyHostToDevice));
    dim3 threads(32,32);
    float4 defVar;
    defVar.x=defVar.y=defVar.z=defVar.w=0;
    dim3 grid(divUp(out_cols, threads.x), divUp(out_rows, threads.y));
    renderFramesKernel<<<grid,threads>>>(rgbIn.rows,rgbIn.cols,rgbIn.data,
                                        out_rows,out_cols,outDataGPU,freshesGPU,
                                        defVar,invGPU,centersGPU,eleNum);
    checkCudaErrors(cudaFree(invGPU));
    checkCudaErrors(cudaFree(outDataGPU));
    checkCudaErrors(cudaFree(freshesGPU));
    checkCudaErrors(cudaFree(centersGPU));
    return true;
}

#endif
